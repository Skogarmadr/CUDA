#include "hip/hip_runtime.h"
#include "Montecarlo.h"

#include <iostream>
#include "MontecarloMultiGPU.h"
#include "Montecarlo.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void montecarlo(hiprandState* ptrDevGenerator, int* ptrDevNbTotalGM, int nbDartByThread); // déclarer dans un autre fichier, kernel coté device
extern __global__ void createGenerator (hiprandState* tabDevGenerator, int deviceId);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

MontecarloMultiGPU::MontecarloMultiGPU(const Grid& grid, int nbDartTotal) :
	grid(grid)
    {

    this->result = 0.0;
    this->nbDartTotal = INT_MAX;
    this->nbDartByDevice = nbDartTotal / Device::getDeviceCount();

    }

//Chaque fois qu il y a un malloc dans un concstructeur(forcément fait dans le consctructeur, il faut un free
MontecarloMultiGPU::~MontecarloMultiGPU(void)
    {

    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void MontecarloMultiGPU::run()
    {
    int nbDevice = Device::getDeviceCount();
    long sum;

#pragma omp parallel for reduction(+:sum)
    for (int deviceId = 0; deviceId < nbDevice; deviceId++)
	{
	Device::setDevice(deviceId);
	Montecarlo montecarlo(grid, nbDartByDevice);
	montecarlo.run();
	sum += montecarlo.getNbDartUnderCurve();
	//cout << sum << endl;
	//cout << nbDartTotal << endl;

	}
	result = 4.0 * sum / nbDartTotal;
    }

float MontecarloMultiGPU::getResult()
    {
    return this->result;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
