#include "hip/hip_runtime.h"
#include "Slice.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void slice(float* ptrTabDev, int nbSlice); // déclarer dans un autre fichier, kernel coté device

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Slice::Slice(const Grid& grid, int nbSlice) :
	nbSlice(nbSlice)
    {

    this->result = 0.0;
    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    this->sizeOctetGM = sizeof(float); // octet
    this->sizeOctetSM = sizeof(float) * this->db.x;
    // MM
	{

	// MM (malloc Device)
	    {
	    //Initialization à zéro
	    Device::malloc(&ptrTabDev, sizeOctetGM);
	    Device::memclear(ptrTabDev, sizeOctetGM);
	    }

	Device::lastCudaError("Slice MM (end allocation)"); // temp debug, facultatif
	}

    }

//Chaque fois qu il y a un malloc dans un concstructeur(forcément fait dans le consctructeur, il faut un free
Slice::~Slice(void)
    {
//MM (device free)
	{
	Device::free(ptrTabDev);
	Device::lastCudaError("Slice MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Slice::run()
    {
    Device::lastCudaError("slice (before)"); // temp debug
    slice<<<dg,db, sizeOctetSM>>>(ptrTabDev, nbSlice); // appel d'un kernel -> assynchrone
    Device::lastCudaError("slice (after)"); // temp debug

    Device::synchronize(); // Temp,debug, only for printf in  GPU, synchronisation explicit

// MM (Device -> Host)
	{
	Device::memcpyDToH(&result, ptrTabDev, sizeOctetGM); // barriere synchronisation implicite
	}
    }

float Slice::getResult()
    {
    return result;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
