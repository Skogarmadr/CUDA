#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <reductionADD.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarlo(hiprandState* ptrDevGeneratorGM, int* ptrDevNxTotalGM, int nbDartByThread);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
static __device__ void reductionIntraThread(int* tabSM, hiprandState* ptrDevGeneratorGM, int nbDartByThread);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void montecarlo(hiprandState* ptrDevGeneratorGM, int* ptrDevNTotalGM, int nbDartByThread)
    {
    extern __shared__ int tabSM[];
    //Reduction intra-thread
    reductionIntraThread(tabSM, ptrDevGeneratorGM, nbDartByThread);
    __syncthreads();
    //Reduction interblock
    reductionADD<int>(tabSM, ptrDevNTotalGM);

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void reductionIntraThread(int* tabSM, hiprandState* ptrDevGeneratorGM, int nbDartByThread)
    {
    const int TID = Indice1D::tid();
    //TODO pattern entrelacement

    const int TID_LOCAL = Indice1D::tidLocal();
    hiprandState localGenerator = ptrDevGeneratorGM[TID];
    int x = 0;
    float xAlea;
    float yAlea;
    for (int i = 1; i <= nbDartByThread; i++)
	{
	xAlea = hiprand_uniform(&localGenerator);
	yAlea = hiprand_uniform(&localGenerator);
	x += (int) (1.f / (1.f + xAlea * xAlea) >= yAlea);
	}
    ptrDevGeneratorGM[TID] = localGenerator;
    tabSM[TID_LOCAL] = x;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

