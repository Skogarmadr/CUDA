#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"


#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void montecarlo(float* ptrTabDev, int nb);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float fpi(float x);
static __device__ void reductionIntraThread(float* tabSM, int nb);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void montecarlo(float* ptrTabDev, int nbSlice)
    {
    extern __shared__ float tabSM[];
    //Reduction intra-thread
    reductionIntraThread(tabSM, nbSlice);
    __syncthreads();
    //Reduction interblock
    reductionADD<float>(tabSM, ptrTabDev);

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void reductionIntraThread(float* tabSM, int nbSlice)
    {
    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    //TODO pattern entrelacement

    const int TID_LOCAL = Indice1D::tidLocal();
    const float DX = 1.0f / (float) nbSlice;
    float sumThread = 0;

    int s = TID;
    while (s < nbSlice)
	{
	float x = s * DX;
	sumThread += fpi(x);
	s += NB_THREAD;
	}

    tabSM[TID_LOCAL] = 4 * sumThread * DX;
    }

__device__ float fpi(float x)
    {
    return 1 / (1.0 + x * x);
    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

