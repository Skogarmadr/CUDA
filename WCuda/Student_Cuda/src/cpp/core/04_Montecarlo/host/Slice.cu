#include "hip/hip_runtime.h"
#include "Montecarlo.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void montecarlo(float* ptrTabDev, int nbMontecarlo); // déclarer dans un autre fichier, kernel coté device

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(const Grid& grid, int nbMontecarlo) :
	nbMontecarlo(nbMontecarlo)
    {

    this->result = 0.0;
    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    this->sizeOctetGM = sizeof(float); // octet
    this->sizeOctetSM = sizeof(float) * this->db.x;
    // MM
	{

	// MM (malloc Device)
	    {
	    //Initialization à zéro
	    Device::malloc(&ptrTabDev, sizeOctetGM);
	    Device::memclear(ptrTabDev, sizeOctetGM);
	    }

	Device::lastCudaError("Montecarlo MM (end allocation)"); // temp debug, facultatif
	}

    }

//Chaque fois qu il y a un malloc dans un concstructeur(forcément fait dans le consctructeur, il faut un free
Montecarlo::~Montecarlo(void)
    {
//MM (device free)
	{
	Device::free(ptrTabDev);
	Device::lastCudaError("Montecarlo MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Montecarlo::run()
    {
    Device::lastCudaError("Montecarlo (before)"); // temp debug
    montecarlo<<<dg,db, sizeOctetSM>>>(ptrTabDev, nbSlice); // appel d'un kernel -> assynchrone
    Device::lastCudaError("slice (after)"); // temp debug

    Device::synchronize(); // Temp,debug, only for printf in  GPU, synchronisation explicit

// MM (Device -> Host)
	{
	Device::memcpyDToH(&result, ptrTabDev, sizeOctetGM); // barriere synchronisation implicite
	}
    }

float Slice::getResult()
    {
    return result;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
