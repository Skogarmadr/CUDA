#include "hip/hip_runtime.h"
#include "Montecarlo.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void montecarlo(hiprandState* ptrDevGenerator, int* ptrDevNbTotalGM, int nbDartByThread); // déclarer dans un autre fichier, kernel coté device
extern __global__ void createGenerator (hiprandState* tabDevGenerator, int deviceId);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(const Grid& grid, int nbDartTotal)
    {

    this->result = 0.0;
    this->nbDartUnderCurve = 0;
    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    this->sizeOctetGM = sizeof(int); // octet
    this->sizeOctetSM = sizeof(int) * this->db.x;
    this->sizeOctetGeneratorGM = grid.threadCounts() * sizeof(hiprandState);
    // MM
	{

	// MM (malloc Device)
	    {
	    //Initialization à zéro
	    Device::malloc(&ptrDevGMResult, sizeOctetGM);
	    Device::memclear(ptrDevGMResult, sizeOctetGM);

	    Device::malloc(&tabDevGeneratorGM, sizeOctetGeneratorGM);
	    Device::memclear(tabDevGeneratorGM, sizeOctetGeneratorGM);
	    }

	Device::lastCudaError("Montecarlo MM (end allocation)"); // temp debug, facultatif
	}
	const int DEVICE_ID = Device::getDeviceId();
	createGenerator<<<dg,db>>>(tabDevGeneratorGM, DEVICE_ID);
	this->nbDartByThread = nbDartTotal / grid.threadCounts();
	this->nbDartTotal = grid.threadCounts() * nbDartByThread;

    }

//Chaque fois qu il y a un malloc dans un concstructeur(forcément fait dans le consctructeur, il faut un free
Montecarlo::~Montecarlo(void)
    {
//MM (device free)
	{
	Device::free(ptrDevGMResult);
	Device::free(tabDevGeneratorGM);
	Device::lastCudaError("Montecarlo MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Montecarlo::run()
    {
    Device::lastCudaError("Montecarlo (before)"); // temp debug
    montecarlo<<<dg,db, sizeOctetSM>>>(tabDevGeneratorGM, ptrDevGMResult, nbDartByThread); // appel d'un kernel -> assynchrone
    Device::lastCudaError("Montecarlo (after)"); // temp debug

    Device::synchronize(); // Temp,debug, only for printf in  GPU, synchronisation explicit
    int resultDevice = 0;
// MM (Device -> Host)
	{

	Device::memcpyDToH(&resultDevice, ptrDevGMResult, sizeOctetGM); // barriere synchronisation implicite
	}
	this->nbDartUnderCurve = resultDevice;
	this->result = 4.0 * resultDevice / nbDartTotal;
    }

float Montecarlo::getResult()
    {
    return this->result;
    }

int Montecarlo::getNbDartUnderCurve()
    {
return this->nbDartUnderCurve;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
