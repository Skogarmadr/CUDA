#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "RayTracingMath.h"

#include "IndiceTools_GPU.h"
#include "Sphere.h"

#include "length_cm.h"

using namespace gpu;

// Attention : 	Choix du nom est important!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ void uploadGPU(Sphere* tabValue);

__global__ void raytracing_GM(Sphere* ptrDevTabSphere, int nbSphere, uchar4* ptrDevPixels, uint w, uint h, float t);
__global__ void raytracing_CM(uchar4* ptrDevPixels, uint w, uint h, float t);

__device__ void work(Sphere* ptrDevTabSphere, int nbSphere, uchar4* ptrDevPixels, uint w, uint h, float t);

// Déclaration Constante globale
__constant__ Sphere TAB_CM[LENGTH_CM];
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
/**
 * call once by the host
 *
 */
__global__ void raytracing_GM(Sphere* ptrDevTabSphere, int nbSphere, uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    work(ptrDevTabSphere, nbSphere, ptrDevPixels, w, h, t);
    }

__global__ void raytracing_CM(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    work(TAB_CM, LENGTH_CM, ptrDevPixels, w, h, t);
    }

__host__ void uploadGPU(Sphere * tabSphere)
    {
    size_t size = LENGTH_CM * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), tabSphere, size, offset, hipMemcpyHostToDevice));
    }

__device__ void work(Sphere* ptrDevTabSphere, int nbSphere, uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    RayTracingMath rayTracingMath = RayTracingMath(nbSphere, ptrDevTabSphere);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i, j;
    int W = w * h;

    int s = TID;
    while (s < W)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	rayTracingMath.colorIJ(&s[ptrDevPixels], i, j, t);

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

